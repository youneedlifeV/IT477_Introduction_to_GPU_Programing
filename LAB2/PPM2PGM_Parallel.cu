#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>

typedef struct
{
    unsigned char red, green, blue;
} PPMPixel;

typedef struct
{
    unsigned char gray;
} PGMPixel;

typedef struct
{
    int x, y;
    PPMPixel *data;
} PPMImage;

typedef struct
{
    int x, y;
    PGMPixel *data;
} PGMImage;

#define CREATOR "V"
#define RGB_COMPONENT_COLOR 255

static PPMImage *readPPM(const char *filename)
{

    char buff[16];
    PPMImage *img;
    FILE *fp;
    int c, rgb_comp_color;
    //open PPM file for reading
    fp = fopen(filename, "rb");
    if (!fp)
    {
        fprintf(stderr, "Unable to open file '%s'\n", filename);
        exit(1);
    }

    //read image format
    if (!fgets(buff, sizeof(buff), fp))
    {
        perror(filename);
        exit(1);
    }

    //check the image format
    if (buff[0] != 'P' || buff[1] != '6')
    {
        fprintf(stderr, "Invalid image format (must be 'P6')\n");
        exit(1);
    }

    //alloc memory form image
    img = (PPMImage *)malloc(sizeof(PPMImage));
    if (!img)
    {
        fprintf(stderr, "Unable to allocate memory\n");
        exit(1);
    }

    //check for comments
    c = getc(fp);
    while (c == '#')
    {
        while (getc(fp) != '\n')
            ;
        c = getc(fp);
    }

    ungetc(c, fp);
    //read image size information
    if (fscanf(fp, "%d %d", &img->x, &img->y) != 2)
    {
        fprintf(stderr, "Invalid image size (error loading '%s')\n", filename);
        exit(1);
    }
    //read rgb component
    if (fscanf(fp, "%d", &rgb_comp_color) != 1)
    {
        fprintf(stderr, "Invalid rgb component (error loading '%s')\n", filename);
        exit(1);
    }

    //check rgb component depth
    if (rgb_comp_color != RGB_COMPONENT_COLOR)
    {
        fprintf(stderr, "'%s' does not have 8-bits components\n", filename);
        exit(1);
    }

    while (fgetc(fp) != '\n')
        ;
    //memory allocation for pixel data
    img->data = (PPMPixel *)malloc(img->x * img->y * sizeof(PPMPixel));

    if (!img)
    {
        fprintf(stderr, "Unable to allocate memory\n");
        exit(1);
    }

    //read pixel data from file
    if (fread(img->data, 3 * img->x, img->y, fp) != img->y)
    {
        fprintf(stderr, "Error loading image '%s'\n", filename);
        exit(1);
    }
    fclose(fp);
    return img;
}
void writePGM(const char *filename, PGMImage *gry)
{
    FILE *fp;
    //open file for output
    fp = fopen(filename, "wb");
    if (!fp)
    {
        fprintf(stderr, "Unable to open file '%s'\n", filename);
        exit(1);
    }
    
    //write the header file
    //image format
    fprintf(fp, "P5\n");
    
    //comments
    fprintf(fp, "# Created by %s\n", CREATOR);
    //image size
    fprintf(fp, "%d %d\n", gry->x, gry->y);
    // rgb component depth
    fprintf(fp, "%d\n", RGB_COMPONENT_COLOR);

    // pixel data
    fwrite(gry->data, gry->x, gry->y, fp);
    fclose(fp);
}

static PGMImage *changeColorPPM(PPMImage *img, double* ttt)
{
    PGMImage *gry;
    int i, grayValue;
    double graymapval;
    if (img)
    {
        gry = (PGMImage *)malloc(sizeof(PGMImage));
        gry->x = img->x;
        gry->y = img->y;
        gry->data = (PGMPixel *)malloc(gry->x * gry->y * sizeof(PGMPixel));
        clock_t start = clock();
        
        for (i = 0; i < img->x * img->y; i++)
        {
            graymapval = 0.3 * (img->data[i].red) + 0.59 * (img->data[i].green) + 0.11 * (img->data[i].blue);
            grayValue = (unsigned char)(((unsigned int)graymapval) % (RGB_COMPONENT_COLOR + 1));
            gry->data[i].gray = grayValue;
        }
        clock_t end = clock();
        *ttt = (((double)(end - start))/CLOCKS_PER_SEC)*1000;
    }
    return gry;
}

__global__ void toGRAY(unsigned char* R, unsigned char* G, unsigned char* B, unsigned char* GRAY, long long n){
        int tid = blockIdx.x*blockDim.x + threadIdx.x;
        if(tid<n) {
                GRAY[tid] = (unsigned char)(((unsigned int)(0.3 * (R[tid]) + 0.59 * (G[tid]) + 0.11 * (B[tid]))) % (256));
            
        }
}

int main()
{       
    char nameppm[] = "JX0.ppm";
    char namepgm[] = "JX0.pgm";
    double serialTime[10];
    double ttt;
    printf("Serial Computation Time: \n");
    printf("I\tTime\n");
    for (int i = 0; i <= 9;i++)
{
    char str[10];
    sprintf(str, "%d", i);
    nameppm[1] = i + '0';
    namepgm[1] = i + '0';
    PPMImage *image;
    PGMImage *grayImage;
    image = readPPM(nameppm);
    double ttt;
    grayImage = changeColorPPM(image,&ttt);
    printf("%d\t%f\n",i,ttt);
    serialTime[i] = ttt;
    writePGM(namepgm, grayImage);
    free(image->data);
    free(image);
    free(grayImage->data);
    free(grayImage);
}
    long long len;
        long long lenb, mx, mn;
        mx = pow(2,10);
        mn = pow(2,0);
        lenb = mn;
    double getTime[11][12];
    getTime[0][0]=0;
    int ii=0;
    for(; lenb<=mx; lenb*=2) {
        
    for (int i = 0; i <= 9;i++) {
        unsigned char *R,*G,*B,*GRAY,*dR,*dG,*dB,*dGRAY;
        char str[10];
        sprintf(str, "%d", i);
        nameppm[1] = i + '0';
        namepgm[1] = i + '0';
        PPMImage *img;
        PGMImage *gry;

        img = readPPM(nameppm);
        len = (img->x * img->y);
        gry = (PGMImage *)malloc(sizeof(PGMImage));
        gry->x = img->x;
        gry->y = img->y;
        gry->data = (PGMPixel *)malloc(gry->x * gry->y * sizeof(PGMPixel));

        R = (unsigned char*)malloc(len*sizeof(unsigned char));
        G = (unsigned char*)malloc(len*sizeof(unsigned char));
        B = (unsigned char*)malloc(len*sizeof(unsigned char));
        GRAY = (unsigned char*)malloc(len*sizeof(unsigned char));
        hipMalloc((void**)&dR,len*sizeof(unsigned char));
        hipMalloc((void**)&dG,len*sizeof(unsigned char));
        hipMalloc((void**)&dB,len*sizeof(unsigned char));
        hipMalloc((void**)&dGRAY,len*sizeof(unsigned char));
        for(int j=0; j<len; j++) {
            R[j] = img->data[j].red;
            G[j] = img->data[j].green;
            B[j] = img->data[j].blue;
        }
        hipMemcpy(dR,R,len*sizeof(unsigned char),hipMemcpyHostToDevice);
        hipMemcpy(dG,G,len*sizeof(unsigned char),hipMemcpyHostToDevice);
        hipMemcpy(dB,B,len*sizeof(unsigned char),hipMemcpyHostToDevice);
        int blockSize = lenb;
        int numBlocks = (len + blockSize -1) / blockSize;
        hipEvent_t start,stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start);

        toGRAY<<<numBlocks, blockSize>>>(dR,dG,dB,dGRAY,len);

        hipEventRecord(stop);
        hipEventSynchronize(stop);
        float tot_time=0;
        hipEventElapsedTime(&tot_time,start,stop);
        getTime[i+1][0] = i;
        getTime[0][ii+1] = lenb; 
        getTime[i+1][ii+1] = tot_time; //FOR PARALLEL TIME GRAPH
        //getTime[i+1][ii+1] = (serialTime[i]/tot_time);  //FOR SPEEDUP GRAPH
        
        hipMemcpy(GRAY,dGRAY,len*sizeof(unsigned char),hipMemcpyDeviceToHost);
        for(int j=0; j<len; j++) {
            gry->data[j].gray = GRAY[j];
        }
        writePGM(namepgm, gry);
        free(R);
        free(G);
        free(B);
        free(GRAY);
        hipFree(dR);
        hipFree(dG);
        hipFree(dB);
        hipFree(dGRAY);
        free(img->data);
        free(img);
        free(gry->data);
        free(gry);

    }
ii++;
}
    printf("Parallel Computation Time: \n");
    for(int i=0; i<11; i++) {
        for(int j=0;j<12;j++) {
            if( j==0 ){
                printf("%.0f", getTime[i][j]);
            }
            else if( i==0 ){
                printf(",%.0f", getTime[i][j]);
            }
            else printf(",%f", getTime[i][j]);
        }
        printf("\n");
    }
    printf("Speed Up: \n");
    for(int i=0; i<11; i++) {
        for(int j=0;j<12;j++) {
            if( j==0 ){
                printf("%.0f", getTime[i][j]);
            }
            else if( i==0 ){
                printf(",%.0f", getTime[i][j]);
            }
            else printf(",%f", (serialTime[i-1]/(double)getTime[i][j]));
        }
        printf("\n");
    }
    return 0;
}
