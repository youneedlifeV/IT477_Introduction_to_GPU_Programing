#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>

typedef struct
{
    unsigned char red, green, blue;
} PPMPixel;

typedef struct
{
    unsigned char gray;
} PGMPixel;

typedef struct
{
    int x, y;
    PPMPixel *data;
} PPMImage;

typedef struct
{
    int x, y;
    PGMPixel *data;
} PGMImage;

#define CREATOR "V"
#define RGB_COMPONENT_COLOR 255

static PPMImage *readPPM(const char *filename)
{

    char buff[16];
    PPMImage *img;
    FILE *fp;
    int c, rgb_comp_color;
    //open PPM file for reading
    fp = fopen(filename, "rb");
    if (!fp)
    {
        fprintf(stderr, "Unable to open file '%s'\n", filename);
        exit(1);
    }

    //read image format
    if (!fgets(buff, sizeof(buff), fp))
    {
        perror(filename);
        exit(1);
    }

    //check the image format
    if (buff[0] != 'P' || buff[1] != '6')
    {
        fprintf(stderr, "Invalid image format (must be 'P6')\n");
        exit(1);
    }

    //alloc memory form image
    img = (PPMImage *)malloc(sizeof(PPMImage));
    if (!img)
    {
        fprintf(stderr, "Unable to allocate memory\n");
        exit(1);
    }

    //check for comments
    c = getc(fp);
    while (c == '#')
    {
        while (getc(fp) != '\n')
            ;
        c = getc(fp);
    }

    ungetc(c, fp);
    //read image size information
    if (fscanf(fp, "%d %d", &img->x, &img->y) != 2)
    {
        fprintf(stderr, "Invalid image size (error loading '%s')\n", filename);
        exit(1);
    }
    //read rgb component
    if (fscanf(fp, "%d", &rgb_comp_color) != 1)
    {
        fprintf(stderr, "Invalid rgb component (error loading '%s')\n", filename);
        exit(1);
    }

    //check rgb component depth
    if (rgb_comp_color != RGB_COMPONENT_COLOR)
    {
        fprintf(stderr, "'%s' does not have 8-bits components\n", filename);
        exit(1);
    }

    while (fgetc(fp) != '\n')
        ;
    //memory allocation for pixel data
    img->data = (PPMPixel *)malloc(img->x * img->y * sizeof(PPMPixel));

    if (!img)
    {
        fprintf(stderr, "Unable to allocate memory\n");
        exit(1);
    }

    //read pixel data from file
    if (fread(img->data, 3 * img->x, img->y, fp) != img->y)
    {
        fprintf(stderr, "Error loading image '%s'\n", filename);
        exit(1);
    }
    fclose(fp);
    return img;
}
void writePGM(const char *filename, PGMImage *gry)
{
    FILE *fp;
    //open file for output
    fp = fopen(filename, "wb");
    if (!fp)
    {
        fprintf(stderr, "Unable to open file '%s'\n", filename);
        exit(1);
    }
    
    //write the header file
    //image format
    fprintf(fp, "P5\n");
    
    //comments
    fprintf(fp, "# Created by %s\n", CREATOR);
    //image size
    fprintf(fp, "%d %d\n", gry->x, gry->y);
    // rgb component depth
    fprintf(fp, "%d\n", RGB_COMPONENT_COLOR);

    // pixel data
    fwrite(gry->data, gry->x, gry->y, fp);
    fclose(fp);
}

__global__ void toGRAY(unsigned char* R, unsigned char* G, unsigned char* B, unsigned char* GRAY, long long n){
        int tid = blockIdx.x*blockDim.x + threadIdx.x;
        if(tid<n) {
                GRAY[tid] = (unsigned char)(((unsigned int)(0.3 * (R[tid]) + 0.59 * (G[tid]) + 0.11 * (B[tid]))) % (256));
            
        }
}

int main()
{       
    char nameppm[] = "JX0.ppm";
    char namepgm[] = "JX0.pgm";
    long long len;
    for (int i = 0; i <= 9;i++) {
        unsigned char *R,*G,*B,*GRAY,*dR,*dG,*dB,*dGRAY;
        char str[10];
        sprintf(str, "%d", i);
        nameppm[1] = i + '0';
        namepgm[1] = i + '0';
        PPMImage *img;
        PGMImage *gry;

        img = readPPM(nameppm);
        len = (img->x * img->y);
        gry = (PGMImage *)malloc(sizeof(PGMImage));
        gry->x = img->x;
        gry->y = img->y;
        gry->data = (PGMPixel *)malloc(gry->x * gry->y * sizeof(PGMPixel));

        R = (unsigned char*)malloc(len*sizeof(unsigned char));
        G = (unsigned char*)malloc(len*sizeof(unsigned char));
        B = (unsigned char*)malloc(len*sizeof(unsigned char));
        GRAY = (unsigned char*)malloc(len*sizeof(unsigned char));
        hipMalloc((void**)&dR,len*sizeof(unsigned char));
        hipMalloc((void**)&dG,len*sizeof(unsigned char));
        hipMalloc((void**)&dB,len*sizeof(unsigned char));
        hipMalloc((void**)&dGRAY,len*sizeof(unsigned char));
        for(int j=0; j<len; j++) {
            R[j] = img->data[j].red;
            G[j] = img->data[j].green;
            B[j] = img->data[j].blue;
        }
        hipMemcpy(dR,R,len*sizeof(unsigned char),hipMemcpyHostToDevice);
        hipMemcpy(dG,G,len*sizeof(unsigned char),hipMemcpyHostToDevice);
        hipMemcpy(dB,B,len*sizeof(unsigned char),hipMemcpyHostToDevice);
        int blockSize = 512;
        int numBlocks = (len + blockSize -1) / blockSize;
        hipEvent_t start,stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start);

        toGRAY<<<numBlocks, blockSize>>>(dR,dG,dB,dGRAY,len);

        hipEventRecord(stop);
        hipEventSynchronize(stop);
        float tot_time=0;
        hipEventElapsedTime(&tot_time,start,stop);
        printf("%d\t%f\n",i,tot_time);
        hipMemcpy(GRAY,dGRAY,len*sizeof(unsigned char),hipMemcpyDeviceToHost);
        for(int j=0; j<len; j++) {
            gry->data[j].gray = GRAY[j];
        }
        writePGM(namepgm, gry);
        free(R);
        free(G);
        free(B);
        free(GRAY);
        hipFree(dR);
        hipFree(dG);
        hipFree(dB);
        hipFree(dGRAY);
        free(img->data);
        free(img);
        free(gry->data);
        free(gry);
    }
    return 0;
}
